#include "hip/hip_runtime.h"
int gCellReso = 1;
int* gCellBeg = NULL;
int* gCellEnd = NULL;
int* gSampleKeys = NULL;

__global__ void UpdateKeys(float2* iData, int iSize, int* oKeys, int iCellReso);

__global__ 
void FindCellStartEnd(int* iKeys,
					  int iKeyNum,
					  int* oCellStart,
					  int* oCellEnd)
{
	extern __shared__ int sharedHash[];    // blockSize + 1 elements
    int index = blockIdx.x*blockDim.x + threadIdx.x;
	
    int key;
    // handle case when no. of particles not multiple of block size
    if (index < iKeyNum) {
        key = iKeys[index];

        // Load hash data into shared memory so that we can look 
        // at neighboring particle's hash value without loading
        // two hash values per thread
	    sharedHash[threadIdx.x+1] = key;

	    if (index > 0 && threadIdx.x == 0)
	    {
		    // first thread in block must load neighbor particle hash
		    sharedHash[0] = iKeys[index-1];
	    }
	}

	__syncthreads();

	if (index < iKeyNum) {
		// If this particle has a different cell index to the previous
		// particle then it must be the first particle in the cell,
		// so store the index of this particle in the cell. 
		// As it isn't the first particle, it must also be the cell end of
		// the previous particle's cell

	    if (index == 0 || key != sharedHash[threadIdx.x])
	    { 
		    oCellStart[key] = index;
            if (index > 0)
                oCellEnd[sharedHash[threadIdx.x]] = index;
	    }

        if (index == iKeyNum - 1)
        {
            oCellEnd[key] = index + 1;
        }
	}
}

int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

void InitGrid(int iSampleSize)
{
	hipMalloc((void**)&gCellBeg, gCellReso*gCellReso*sizeof(int));
	hipMalloc((void**)&gCellEnd, gCellReso*gCellReso*sizeof(int));
	hipMalloc((void**)&gSampleKeys, iSampleSize*sizeof(int));
}

__device__ __host__
int CalcCellIDInt(int iCellX, int iCellY, int iCellReso)
{
	return iCellX+iCellY*iCellReso;
}

__device__ __host__
int CalcCellID(float ix, float iy, int iCellReso)
{
	//ix = floor(ix*iCellReso);
	//iy = floor(iy*iCellReso);
	//int x = ix;
	//int y = iy;
	//return CalcCellIDInt(x, y, iCellReso);
	return CalcCellIDInt(int(floor(ix*iCellReso)), int(floor(iy*iCellReso)), iCellReso);
}

void BuildGrid(float2* ioData,
			   int* ioKeys,
			   int iDataNum,
			   int* oCellStart,
			   int* oCellEnd
			   )
{
    int numThreads, numBlocks;
    computeGridSize(iDataNum, 256, numBlocks, numThreads);
	
	UpdateKeys<<<numBlocks, numThreads>>>(ioData, iDataNum, ioKeys, gCellReso);

	int celNum = gCellReso*gCellReso;
	thrust::sort_by_key(
		thrust::device_ptr<int>(ioKeys),
		thrust::device_ptr<int>(ioKeys+iDataNum),
		thrust::device_ptr<float2>(ioData));
	hipMemset(oCellStart, 0, celNum*sizeof(int));
	hipMemset(oCellEnd, 0, celNum*sizeof(int));

    int smemSize = sizeof(int)*(numThreads+1);
	FindCellStartEnd<<<numBlocks, numThreads, smemSize>>>
		(ioKeys, iDataNum, oCellStart, oCellEnd);

	//thrust::host_vector<int> cellStart(thrust::device_ptr<int>(oCellStart),
	//	thrust::device_ptr<int>(oCellStart+gCellReso*gCellReso));
	//thrust::host_vector<int> cellEnd(thrust::device_ptr<int>(oCellEnd),
	//	thrust::device_ptr<int>(oCellEnd+gCellReso*gCellReso));
	//for (int checkI = 0; checkI < gCellReso*gCellReso; checkI++)
	//{
	//	thrust::host_vector<float2> data(thrust::device_ptr<float2>(ioData+cellStart[checkI]),
	//		thrust::device_ptr<float2>(ioData+cellEnd[checkI]-1));
	//	for (int i=0; i<data.size(); i++)
	//		if (CalcCellID(data[i].x, data[i].y, gCellReso)!=checkI)
	//			printf("!");
	//}
	////for (int i=0; i<cellStart.size(); i++)
	////	printf("(%d,%d)\t", cellStart[i], cellEnd[i]);
	////printf("\n");
	//printf("\n");
}

__device__ __host__
int CalcCellBegY(float iy, float iDDARange, int iCellReso)
{
	//return 0;
	return int(floor((iy-iDDARange)*iCellReso));
}

__device__ __host__
int CalcCellEndY(float iy, float iDDARange, int iCellReso)
{
	return int(ceil((iy+iDDARange)*iCellReso));
}

__device__
int CalcCellBegX(float ix, float iy, int iCellY, float iDDARange, int iCellReso)
{
	//return 0;
	float yup = iy-float(iCellY)/iCellReso;
	float ydown = iy-float(iCellY+1)/iCellReso;
	float ylen = min(fabs(yup), fabs(ydown));
	if (yup*ydown<=0)
		ylen = 0;
	float dx = sqrt(iDDARange*iDDARange-ylen*ylen);
	return int( floor((ix-dx)*iCellReso) );
}

__device__
int CalcCellEndX(float ix, float iy, int iCellY, float iDDARange, int iCellReso)
{
	//return iCellReso-1;
	float yup = iy-float(iCellY)/iCellReso;
	float ydown = iy-float(iCellY+1)/iCellReso;
	float ylen = min(fabs(yup), fabs(ydown));
	if (yup*ydown<=0)
		ylen = 0;
	float dx = sqrt(iDDARange*iDDARange-ylen*ylen);
	return int( ceil((ix+dx)*iCellReso) );
}

__device__
int wrap(int iVal, int iHighBound)
{
	int retVal = iVal - iHighBound*(iVal/iHighBound);
	retVal += iHighBound;
	return retVal%iHighBound;
}

__global__ void DDAGrid(float2* iData,
					  int iDataSize,
					  int iCellReso,
					  int* iCellIndexBeg,
					  int* iCellIndexEnd,
					  float* oHist,
					  int iHistReso,
					  float iDDARange)
{
	unsigned long valindex = blockIdx.x * 256 + threadIdx.x;
	if(valindex<iDataSize)
	{
		float2 pst = iData[valindex];
		float dataX = pst.x;
		float dataY = pst.y;
		int cellYBeg = CalcCellBegY(dataY, iDDARange, iCellReso);
		int cellYEnd = CalcCellEndY(dataY, iDDARange, iCellReso);
		for (int scellY = cellYBeg; scellY <= cellYEnd; scellY++)
		{
			int cellY = wrap(scellY, iCellReso);
			int cellXBeg = CalcCellBegX(dataX, dataY, scellY, iDDARange, iCellReso);
			int cellXEnd = CalcCellEndX(dataX, dataY, scellY, iDDARange, iCellReso);
			for (int scellX = cellXBeg; scellX <= cellXEnd; scellX++)
			{
				int cellX = wrap(scellX, iCellReso);
				int cellID = CalcCellIDInt(cellX, cellY, iCellReso);
				int begIdx = iCellIndexBeg[cellID];
				int endIdx = iCellIndexEnd[cellID];
				for (int i=begIdx; i<endIdx; i++)
				if (i!=valindex)
				{
					DDACore(dataX, dataY, (float*)iData, oHist, i, iDataSize, iDDARange, iHistReso);
				}
			}
		}
	}
}

__global__ void ChangeAndCount(float *data,float *data_new,size_t size,int iCellReso, int* oCount)
{
	unsigned long valindex = blockIdx.x * 256 + threadIdx.x;
	if(valindex<size)
	{
		float dataOldX = data[valindex*2];
		float dataOldY = data[valindex*2+1];
		float dataNewX = data_new[valindex*2];
		float dataNewY = data_new[valindex*2+1];
		dataNewX -= floor(dataNewX);
		dataNewY -= floor(dataNewY);

		oCount[valindex] =
			(CalcCellID(dataNewX, dataNewY, iCellReso)!=CalcCellID(dataOldX, dataOldY, iCellReso));

		data[valindex*2]=data_new[valindex*2]=dataNewX;
		data[valindex*2+1]=data_new[valindex*2+1]=dataNewY;

	}
}

__global__ void UpdateKeys(float2* iData, int iSize, int* oKeys, int iCellReso)
{
	int valindex = blockIdx.x * 256 + threadIdx.x;
	if (valindex<iSize)
	{
		float2 pst = iData[valindex];
		oKeys[valindex] = CalcCellID(pst.x,
							pst.y,
							iCellReso);
	}
}

__global__ void CalcForceGrid(float2 *iData,
							int iDataSize,
							float2* oForce,
							float* oPower,
							float iDDARange,
							int* iCellIndexBeg,
							int* iCellIndexEnd,
							int iCellReso
								)
{
	unsigned long valindex = blockIdx.x * 256 + threadIdx.x;
	if(valindex<iDataSize)
	{
		float addX = 0;
		float addY = 0;
		float2 pst = iData[valindex];
		float dataX = pst.x;
		float dataY = pst.y;
		int cellYBeg = CalcCellBegY(dataY, iDDARange, iCellReso);
		int cellYEnd = CalcCellEndY(dataY, iDDARange, iCellReso);
		for (int scellY = cellYBeg; scellY <= cellYEnd; scellY++)
		{
			int cellY = wrap(scellY, iCellReso);
			int cellXBeg = CalcCellBegX(dataX, dataY, scellY, iDDARange, iCellReso);
			int cellXEnd = CalcCellEndX(dataX, dataY, scellY, iDDARange, iCellReso);
			for (int scellX = cellXBeg; scellX <= cellXEnd; scellX++)
			{
				int cellX = wrap(scellX, iCellReso);
				int cellID = CalcCellIDInt(cellX, cellY, iCellReso);
				int begIdx = iCellIndexBeg[cellID];
				int endIdx = iCellIndexEnd[cellID];
				for (int i=begIdx; i<endIdx; i++)
				if (i!=valindex)
				{
					CalcMoveLenCore(
						(float*)iData,
						dataX,
						dataY,
						i,
						iDDARange,
						addX,
						addY);
				}
			}
		}
		oForce[valindex] = make_float2(addX, addY);
		oPower[valindex] = addX*addX+addY*addY;
	}
}
