#include "hip/hip_runtime.h"
// includes, system

#define _DER_ADJ

#define _DISTURB
#define _GLOBAL_DISTURB

#define _DISTURB_FACTOR 0.00015f

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hiprand.h>
#include <hip/hip_vector_types.h>

#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "kernel.h"
#include "kernel_b.cu"
#include "BuildGrid.cu"

//#define _NON_WRAP

extern __global__ void MoveAdapt(float *data,float *data_new,size_t size,float DIS,int RESOLUTION,float CONSTANT);
extern __global__ void DDAAdapt(float *data, float *resolution,size_t size,float DIS,int RESOLUTION);

float *dev_data = 0;
float *dev_res = 0;
float *dev_target=0;
float *dev_diff=0;
float *dev_cov=0;
float *dev_cov_x=0;
float *dev_cov_y=0;
float *dev_datanew=0;
float *dev_randNum=0;
float *dev_gauss_coe = 0;
float *dev_gauss_der_coe = 0;
hipArray* cu_array_x;
hipArray* cu_array_y;
bool gbIsAdapt = false;
int gInnerLoop = 10;
hiprandGenerator_t gGen;
float gDelta = 1.5f;
int lRows;
int lCols;
float gAdaptAspect = 1.0f;
bool gbIsChangeStep = true;
bool gIsUseCell = true;
float2 *dev_force=0;
float *dev_force_len=0;
float gGridRelativeSize = 0.1f;

void InitKernel(float* data,
				 float* target,
				 size_t sampleSize,
				 size_t Resolution,
				 int seed,
				 float iDDASize)
{
	hipMalloc((void**)&dev_datanew, sampleSize*2 * sizeof(float));
	hipMemcpy(dev_datanew, data, sampleSize*2 * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_data, sampleSize*2 * sizeof(float));
	hipMemcpy(dev_data, data, sampleSize*2 * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_res,  Resolution*Resolution * sizeof(float));

	hipMalloc((void**)&dev_target,  Resolution*Resolution * sizeof(float));
	hipMemcpy(dev_target, target, Resolution*Resolution* sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_cov,  Resolution*Resolution * sizeof(float));
	hipMalloc((void**)&dev_diff,  Resolution*Resolution * sizeof(float));

	hipMalloc((void**)&dev_cov_x,  Resolution*Resolution* sizeof(float));
	hipMalloc((void**)&dev_cov_y,  Resolution*Resolution* sizeof(float));

	hipMalloc(&dev_randNum, sampleSize*2*sizeof(float));
	hiprandCreateGenerator(&gGen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gGen, seed);

    hipChannelFormatDesc channelDesc_x = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc_y = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMallocArray( &cu_array_x, &channelDesc_x, Resolution,Resolution );
	hipMallocArray( &cu_array_y, &channelDesc_y, Resolution,Resolution );
	tex_x.addressMode[0] = hipAddressModeClamp;
	tex_x.addressMode[1] = hipAddressModeClamp;
	tex_x.filterMode = hipFilterModeLinear;
	tex_x.normalized = true;  
	tex_y.addressMode[0] = hipAddressModeClamp;
	tex_y.addressMode[1] = hipAddressModeClamp;
	tex_y.filterMode = hipFilterModeLinear;
	tex_y.normalized = true;  

	// init gaussian coefficients
	int kernelSize = int(gDelta*3.0f)*2+1;
	hipMalloc((void**)&dev_gauss_coe, kernelSize * kernelSize * sizeof(float));
	hipMalloc((void**)&dev_gauss_der_coe, kernelSize * kernelSize * sizeof(float) * 2);
	_CHECK_ERROR

	const int blockSize = 8;
	dim3 grid ((kernelSize-1)/blockSize+1, (kernelSize-1)/blockSize+1);
	dim3 threads (blockSize, blockSize, 1);
	InitGaussCoe<blockSize><<<grid, threads>>>(dev_gauss_coe, dev_gauss_der_coe, kernelSize, gDelta);
	_CHECK_ERROR

	hipMalloc((void**)&dev_force_len, sampleSize * sizeof(float));
	hipMalloc((void**)&dev_force, sampleSize * sizeof(float2));
	_CHECK_ERROR

	if (gIsUseCell)
	{
		gCellReso = int(1.0f/(gGridRelativeSize*iDDASize));
		if (gCellReso<1)
			gCellReso = 1;
		InitGrid(sampleSize);
		_CHECK_ERROR
		printf("Uniform Grid Resolution %d\n", gCellReso);
	}
}

struct square
{
	__host__ __device__
	float operator()(float x)
	{
		return x * x;
	}
};

float snrm2_fast(float* x, int size)
{
	// with fusion
	return sqrt( thrust::transform_reduce(
		thrust::device_ptr<float>(x),
		thrust::device_ptr<float>(x+size),
		square(),
		0.0f,
		thrust::plus<float>()));
}

float Sampling(float* data,
						 float* res,
						 float* cov,
						 size_t sampleSize,
						 size_t Resolution,
						 float DDA_dis,
						 float step)
{
	dim3 grid ((sampleSize-1)/256+1, 1);
	dim3 threads (256, 1, 1);
	dim3 grid2(( Resolution*Resolution-1)/256+1,1);

	hipChannelFormatDesc channelDesc_x = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc_y = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	
	float sumErr = 0;
	for(int iter=0;iter<gInnerLoop;iter++)
	{
		if (gIsUseCell && !gbIsAdapt)
			BuildGrid((float2*)dev_data, gSampleKeys, sampleSize, gCellBeg, gCellEnd);
		float error = 0;
		hipMemset(dev_res, 0, Resolution*Resolution*sizeof(float));

		_CHECK_ERROR
		if (gbIsAdapt)
			DDAAdapt<<<grid, threads>>>(dev_data, dev_res, sampleSize,DDA_dis,Resolution, gAdaptAspect);
		else
		{
			if (gIsUseCell)
				DDAGrid<<<grid, threads>>>((float2*)dev_data,
					sampleSize,
					gCellReso,
					gCellBeg,
					gCellEnd,
					dev_res,
					Resolution,
					DDA_dis
					);
			else
				DDA<<<grid, threads>>>(dev_data, dev_res, sampleSize,DDA_dis,Resolution);
		}
		_CHECK_ERROR


		Gaussian<<<grid2, threads>>>(dev_res, dev_target,dev_cov, dev_diff,Resolution*Resolution,Resolution,int(gDelta*3.0f),gDelta,dev_gauss_coe);
		_CHECK_ERROR
		error = snrm2_fast(dev_cov, Resolution*Resolution);
		float disturbStep=snrm2_fast(dev_diff, Resolution*Resolution)/error;
		sumErr += error;

        Gaussian2<<<grid2, threads>>>(dev_cov, dev_cov_x,dev_cov_y, Resolution*Resolution,Resolution,int(gDelta*3.0f),gDelta, dev_gauss_der_coe);
		_CHECK_ERROR      
		hipMemcpyToArray(cu_array_x, 0, 0,dev_cov_x, Resolution*Resolution*4, hipMemcpyDeviceToDevice);
	    
		_CHECK_ERROR
		hipMemcpyToArray(cu_array_y, 0, 0,dev_cov_y, Resolution*Resolution*4, hipMemcpyDeviceToDevice);

		hiprandGenerateUniform(gGen, dev_randNum, sampleSize*2);

		hipBindTextureToArray( tex_x, cu_array_x, channelDesc_x);
		hipBindTextureToArray( tex_y, cu_array_y, channelDesc_y);
		if (gbIsAdapt)
			MoveAdapt<<<grid, threads>>>(dev_data, dev_datanew, dev_randNum, sampleSize,DDA_dis, Resolution, step, lRows, lCols, gAdaptAspect);
		else
		{
			if (gIsUseCell)
				CalcForceGrid<<<grid, threads>>>((float2*)dev_data,
													sampleSize,
													dev_force,
													dev_force_len,
													DDA_dis,
													gCellBeg,
													gCellEnd,
													gCellReso);
			else
				CalcForce<<<grid, threads>>>(dev_data,
					dev_force,
					dev_force_len,
					sampleSize,
					DDA_dis,
					Resolution);
			_CHECK_ERROR
			float maxPow = thrust::reduce(thrust::device_ptr<float>(dev_force_len),
				thrust::device_ptr<float>(dev_force_len+sampleSize),
				0.0f,
				thrust::maximum<float>());
			_CHECK_ERROR
			MoveForce<<<grid, threads>>>
				(dev_data,
				dev_datanew,
				sampleSize,
				thrust::raw_pointer_cast(&dev_force[0]),
				step/sqrt(maxPow),
				step*disturbStep*0.02f,
				dev_randNum);
			_CHECK_ERROR
		}
		_CHECK_ERROR
		Change<<<grid, threads>>>(dev_data, dev_datanew,sampleSize);
		_CHECK_ERROR
	}
	hipMemcpy(res, dev_res, Resolution*Resolution * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(cov, dev_cov, Resolution*Resolution * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(data, dev_data, sampleSize*2 * sizeof(float), hipMemcpyDeviceToHost);
	return sumErr/gInnerLoop;
}

void SetAdaptive(float* iAdapt,
				 float* iDAdapt,
				 int iRows,
				 int iCols)
{
	lRows = iRows;
	lCols = iCols;

    hipChannelFormatDesc channelDesc_adapt = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc_dadapt = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
	hipArray* cu_array_adapt;
	hipArray* cu_array_dadapt;

	hipMallocArray( &cu_array_adapt, &channelDesc_adapt, iCols, iRows);
	hipMallocArray( &cu_array_dadapt, &channelDesc_dadapt, iCols, iRows);
	hipMemcpyToArray(cu_array_adapt, 0, 0, iAdapt, iCols*iRows*4, hipMemcpyHostToDevice);
	hipMemcpyToArray(cu_array_dadapt, 0, 0, iDAdapt, iCols*iRows*4*2, hipMemcpyHostToDevice);

	_CHECK_ERROR
	gAdapt.addressMode[0] = hipAddressModeWrap;
	gAdapt.addressMode[1] = hipAddressModeWrap;
	gAdapt.filterMode = hipFilterModeLinear;
	gAdapt.normalized = true;

	gDAdapt.addressMode[0] = hipAddressModeWrap;
	gDAdapt.addressMode[1] = hipAddressModeWrap;
#ifdef _DER_ADJ
	gDAdapt.filterMode = hipFilterModePoint;
#else // _DER_ADJ
	gDAdapt.filterMode = hipFilterModeLinear;
#endif // _DER_ADJ
	gDAdapt.normalized = true;

	hipBindTextureToArray( gAdapt, cu_array_adapt, channelDesc_adapt);
	hipBindTextureToArray( gDAdapt, cu_array_dadapt, channelDesc_dadapt);
	_CHECK_ERROR

}


