#include "hip/hip_runtime.h"
texture<float2, 2, hipReadModeElementType> gDAdapt;
texture<float, 2, hipReadModeElementType> gAdapt;

texture<float, 2, hipReadModeElementType> tex_x;
texture<float, 2, hipReadModeElementType> tex_y;

__global__ void SetZero(float *oData, int iSize)
{
	int x = 256*blockIdx.x+threadIdx.x;
	if (x<iSize)
		oData[x]=0;
}

__device__ void DDACore(float iX,
						float iY,
						float *data,
						float *resolution,
						int cmpIndex,
						size_t size,
						float DIS,
						int RESOLUTION)
{
	float dx=iX-data[cmpIndex*2];
	float dy=iY-data[cmpIndex*2+1];
	dx = dx-floorf(dx+0.5f);
	dy = dy-floorf(dy+0.5f);

	if((fabs(dx)<DIS)&&(fabs(dy)<DIS))
	{
		int index_x=floor(dx*RESOLUTION/2.0f/DIS+RESOLUTION/2);
		int index_y=floor(dy*RESOLUTION/2.0f/DIS+RESOLUTION/2);
		float addVal = (float)1/(float)size;
		atomicAdd(resolution+index_y*RESOLUTION+index_x, addVal);

	}
}

__global__ void DDA(float *data, float *resolution,size_t size,float DIS,int RESOLUTION)
{
	unsigned long valindex = blockIdx.x * 256 + threadIdx.x;
	if(valindex<size)
	{
		float x = data[valindex*2];
		float y = data[valindex*2+1];
		for(int i=0;i<size;i++)
		if(valindex!=i)
		{
			DDACore(x, y, data, resolution, i, size, DIS, RESOLUTION);
		}
	}
//	__syncthreads();

}

__global__ void DDAAdapt(float *data, float *resolution,size_t size,float DIS,int RESOLUTION, float iAspect)
{
	unsigned long valindex = blockIdx.x * 256 + threadIdx.x;
	if(valindex<size)
	{
		float2 pst0 = make_float2(data[valindex*2], data[valindex*2+1]);
		float r0 = tex2D(gAdapt, pst0.x, pst0.y);
		for(int i=0;i<size;i++)
		if(valindex!=i)
		{
			float2 pst1 = make_float2(data[i*2], data[i*2+1]);
			float r1 = tex2D(gAdapt, pst1.x, pst1.y);
			float dx=pst0.x-pst1.x;
			float dy=pst0.y-pst1.y;
#ifndef _NON_WRAP
			dx = dx-floorf(dx+0.5f);
			dy = dy-floorf(dy+0.5f);
#endif

			dy *= iAspect;

			float A = 2.0f/(r0+r1);
			dx *= A;
			dy *= A;

			if((dx*dx<DIS*DIS)&&(dy*dy<DIS*DIS))
			{
				int index_x=dx*RESOLUTION/2.0f/DIS+RESOLUTION/2.0f;
				int index_y=dy*RESOLUTION/2.0f/DIS+RESOLUTION/2.0f;
				float addVal = (float)1/(float)size;
				atomicAdd(resolution+index_y*RESOLUTION+index_x, addVal);
			}
		}
	}
//	__syncthreads();

}

__device__ void CalcMoveLenCore(float *iData,
								float iX,
								float iY,
								int iCmpIndex,
								float iDDARange,
								float& oAddX,
								float& oAddY
								)
{
	float dx_temp=iX-iData[iCmpIndex*2];
	float dy_temp=iY-iData[iCmpIndex*2+1];
	dx_temp = dx_temp-floorf(dx_temp+0.5f);
	dy_temp = dy_temp-floorf(dy_temp+0.5f);

	if((dy_temp*dy_temp+dx_temp*dx_temp)<iDDARange*iDDARange*0.81)
	{
		float u=dx_temp/2/iDDARange+0.5f;
		float v=dy_temp/2/iDDARange+0.5f;

		float mx = tex2D(tex_x, u, v);
		float my = tex2D(tex_y, u, v);

		oAddX += mx;
		oAddY += my;
	}
}

__device__ void CalcMoveLen(float *data,
							int valindex,
							size_t size,
							float DIS,
							int RESOLUTION,
							float& oMoveX,
							float& oMoveY)
{
	float addx = 0;
	float addy = 0;
	float x = data[valindex*2];
	float y = data[valindex*2+1];
	for(int i=0;i<size;i++)
	{   
		if(valindex!=i)
		{
			CalcMoveLenCore(data, x, y, i, DIS, addx, addy);
		}
	}
	oMoveX = addx;
	oMoveY = addy;
}

__global__ void CalcForce(float *data,
						  float2 *oForce,
						  float *oPower,
						  size_t size,
						  float DIS,
						  int RESOLUTION)
{
	unsigned long valindex = blockIdx.x * 256 + threadIdx.x;

	if(valindex<size)
	{
		float addx=0;
		float addy=0;
		CalcMoveLen(data, valindex, size, DIS, RESOLUTION, addx, addy);
		oForce[valindex] = make_float2(addx, addy);
		oPower[valindex] = addx*addx+addy*addy;
	}

}

__global__ void MoveForce(float *data,
						  float *newData,
						  int size,
						  float2 *force,
						  float scale,
						  float disturbScale,
						  float *disturb)
{
	unsigned long valindex = blockIdx.x * 256 + threadIdx.x;

	if(valindex<size)
	{
#ifdef _GLOBAL_DISTURB
		newData[valindex*2] = data[valindex*2]+force[valindex].x*scale+(disturb[valindex*2]-0.5f)*disturbScale;
		newData[valindex*2+1] = data[valindex*2+1]+force[valindex].y*scale+(disturb[valindex*2+1]-0.5f)*disturbScale;
#else
		newData[valindex*2] = data[valindex*2]+force[valindex].x*scale;
		newData[valindex*2+1] = data[valindex*2+1]+force[valindex].y*scale;
#endif
	}
}

__global__ void MoveAdapt(float *data,float *data_new, float* randNum,size_t size,float DIS,int RESOLUTION,float CONSTANT, int iRows, int iCols, float iAspect)
{
	unsigned long valindex = blockIdx.x * 256 + threadIdx.x;

	if(valindex<size)
	{
		float mx = 0;
		float my = 0;
		float2 pst0 = make_float2(data[valindex*2], data[valindex*2+1]);
		float r0 = tex2D(gAdapt, pst0.x, pst0.y);
#ifdef _DER_ADJ
		float2 dR0 = tex2D(gDAdapt, pst0.x+0.5f/iCols, pst0.y+0.5f/iRows);
#else // _DER_ADJ
		float2 dR0 = tex2D(gDAdapt, pst0.x, pst0.y);
#endif // _DER_ADJ
		for(int i=0;i<size;i++)
		{ 
			if(valindex!=i)
			{
				float2 pst1 = make_float2(data[i*2], data[i*2+1]);
				float r1 = tex2D(gAdapt, pst1.x, pst1.y);
				float2 dpst = make_float2(pst1.x-pst0.x, pst1.y-pst0.y);
#ifndef _NON_WRAP
				dpst.x = dpst.x-floorf(dpst.x+0.5f);
				dpst.y = dpst.y-floorf(dpst.y+0.5f);
#endif
				dpst.y *= iAspect;

				float A = 2.0f/(r0+r1);
				float2 scaleDpst = make_float2(dpst.x*A, dpst.y*A);
				float len = sqrt(scaleDpst.x*scaleDpst.x+scaleDpst.y*scaleDpst.y);
				if (len<DIS*0.9f)
				{
					float tmp0 = -2.0f/(r0+r1)/(r0+r1);
					float2 dA = make_float2(tmp0*dR0.x, tmp0*dR0.y);
					float daxx = dA.x*dpst.x-A;
					float dayx = dA.x*dpst.y;
					float dayy = dA.y*dpst.y-A;
					float daxy = dA.y*dpst.x;
					float fx = tex2D(tex_x, scaleDpst.x/DIS/2+0.5f, scaleDpst.y/DIS/2+0.5f);
					float fy = tex2D(tex_y, scaleDpst.x/DIS/2+0.5f, scaleDpst.y/DIS/2+0.5f);

					mx += (fx*daxx+fy*dayx);
					my += (fy*dayy+fx*daxy);
				}

			}
		}
		float moveX;
		float moveY;
#ifdef _GLOBAL_DISTURB
		float randx = randNum[valindex*2]-0.5f;
		float randy = randNum[valindex*2+1]-0.5f;

		moveX = CONSTANT*(mx+randx*_DISTURB_FACTOR/sqrt((float)size))*r0;
		moveY = CONSTANT*(my+randy*_DISTURB_FACTOR/sqrt((float)size))*r0;
#else // _GLOBAL_DISTURB
		moveX = CONSTANT*mx*r0;
		moveY = CONSTANT*my*r0;
#endif // _GLOBAL_DISTURB
		moveY /= iAspect;

		data_new[valindex*2] += moveX;
		data_new[valindex*2+1] += moveY;
	}
}

__global__ void Change(float *data,float *data_new,size_t size)
{
	unsigned long valindex = blockIdx.x * 256 + threadIdx.x;
	if(valindex<size)
	{
		data_new[valindex*2]-=floor(data_new[valindex*2]);
		data_new[valindex*2+1]-=floor(data_new[valindex*2+1]);
		data[valindex*2]=data_new[valindex*2];
		data[valindex*2+1]=data_new[valindex*2+1];

	}
}

__global__ void Gaussian(float* res, float* target,float *cov, float *oDiff,size_t size,int RESOLUTION,int Kernel_size,float Deviation, float* iGaussCoe)
{
    unsigned long valindex = blockIdx.x * 256 + threadIdx.x;
	if(valindex<size)
	{
		int kernelAlign = Kernel_size*2+1;
		cov[valindex]=0;
		int index_x=valindex%RESOLUTION;
		int index_y=valindex/RESOLUTION;
		float weight=0;
		float cmp = RESOLUTION*0.9f/2;
		cmp = cmp*cmp;
		float dx=index_x-RESOLUTION/2.0f+0.5f;
		float dy=index_y-RESOLUTION/2.0f+0.5f;
		if (dx*dx+dy*dy<cmp)
			oDiff[valindex]=res[valindex]-target[valindex];
		else
			oDiff[valindex]=0;
		for(int i=index_x-Kernel_size;i<=index_x+Kernel_size;i++)
		for(int j=index_y-Kernel_size;j<=index_y+Kernel_size;j++)
		{
			int index_new=j*RESOLUTION+i;
			float value=0;
			float x = i-RESOLUTION/2.0f+0.5f;
			float y = j-RESOLUTION/2.0f+0.5f;

			int gaussIndex = (i-index_x+Kernel_size)+(j-index_y+Kernel_size)*kernelAlign;
			float gaussVal = iGaussCoe[gaussIndex];
			if(x*x+y*y<cmp)
			   value=(res[index_new]-target[index_new])*gaussVal;
			cov[valindex]+=value;
			weight+=gaussVal;
		}
		cov[valindex]=cov[valindex]/weight;
	}
}


__global__ void  Gaussian2( float *cov1, float *cov_x, float *cov_y,size_t size,int RESOLUTION,int Kernel_size,float Deviation, float* iGaussDerCoe)
{
	unsigned long valindex = blockIdx.x * 256 + threadIdx.x;
	if(valindex<size)
	{
		int kernelAlign = Kernel_size*2+1;
		cov_x[valindex]=0;
		cov_y[valindex]=0;
		int index_x=valindex%RESOLUTION;
		int index_y=valindex/RESOLUTION;
		for(int i=index_x-Kernel_size;i<=index_x+Kernel_size;i++)
			for(int j=index_y-Kernel_size;j<=index_y+Kernel_size;j++)

			{
				if(i>-1&&j>-1&&i<RESOLUTION&&j<RESOLUTION)
				{
					int index_new=j*RESOLUTION+i;
					int gaussIndex = (i-index_x+Kernel_size)+(j-index_y+Kernel_size)*kernelAlign;
					cov_x[valindex]+=cov1[index_new]*iGaussDerCoe[gaussIndex*2];
					cov_y[valindex]+=cov1[index_new]*iGaussDerCoe[gaussIndex*2+1];
				}

			}
	}
//	__syncthreads();
}

template <int BlockSize>
__global__ void InitGaussCoe(float* oGaussCoe, float* oGaussDerCoe, int iSize, float iSigma)
{
	int i = blockIdx.x*BlockSize+threadIdx.x;
	int j = blockIdx.y*BlockSize+threadIdx.y;
	if (i<iSize && j<iSize)
	{
		float scale = 1.0f/(2*3.141592654f*iSigma*iSigma);

		float x = i-iSize/2.0f+0.5f;
		float y = j-iSize/2.0f+0.5f;

		float gaussVal = scale * exp(-(x*x+y*y)/2.0f/iSigma/iSigma);

		int index = j*iSize+i;
		oGaussCoe[index]=gaussVal;
		oGaussDerCoe[index*2]=-gaussVal*gaussVal*x/iSigma/iSigma;
		oGaussDerCoe[index*2+1]=-gaussVal*gaussVal*y/iSigma/iSigma;
	}
}

